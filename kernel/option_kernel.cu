#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>

#include <hiprand.h>
#include "option_kernel.h"
#include "../cuPrintf.cu"

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cmath>


template<typename T>
struct square
{
    __host__ __device__
        T operator()(const T& x) const{
            return x*x;
        }
};


__device__ float phi(float x) {
    return 0.5*(1 + erf(x/sqrtf(2)));
}

__device__ float get_black_scholes_continuation_value_gpu(float x, float time, int height, InputData indata ) {
    float del_t = indata.expiry_time/(height-1)/365;
    float t = time*del_t;
//    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    float d1, d2, den;
    float ttm = (indata.expiry_time - t)/365;
    d1 = log(x/indata.strike_price) + ( indata.discount_rate + 0.5*indata.volatility*indata.volatility )*ttm;
    d2 = log(x/indata.strike_price) + ( indata.discount_rate - 0.5*indata.volatility*indata.volatility )*ttm;
    den = indata.volatility*sqrtf( ttm );
    d1 = d1/den;
    d2 = d2/den;

    return indata.strike_price*exp(-1*indata.discount_rate*ttm)*phi(-1*d2) - x*phi(-1*d1);
    // cuPrintf("htid[%d] = %f\n", tid, h[tid]);
    //printf("d1: %g, d2: %g, den: %g, phi(-1*d2): %g, phi(-1*d1): %g, h[i]: %g, x[i]: %g\n", d1, d2, den, phi(-1*d2), phi(-1*d1), h[i], x[i]);
}

static __global__ void generate_asset_price_paths_and_cash_flow_reg(float *option_value_eu, float *option_value_am, int width, int height, InputData indata, float *norm_sample) {
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int pathid = blockDim.x * blockIdx.x + threadIdx.x;

	
	int ts_am = height-1;
	int oeb = ts_am;		// optimal_exercise_boundary for this path
	float cf_am = 0;		// cash flow of american option for this path
	
	// the following should be single read and broadcast for all threads I hope
	float spot_price = indata.S_0;
	float strike_price = indata.strike_price;
	float expiry_time = indata.expiry_time;
	float discount_rate = indata.discount_rate;
	float volatility = indata.volatility;
	
	//if ( pathid == 256  )
	//		cuPrintf ("\nAt t=0, spot_price = %g, strike_price = %g\n", spot_price, strike_price);
			
	// for decicing optimal exercise boundary in american options
	float put_value = 0;
	float h = 0;
	float time = 0;
	float d1, d2, den, ttm;
	
	// assuming uniformaly distributed option exercise times
	float drift = discount_rate - indata.dividend - 0.5*pow(volatility,2);
    float del_t = expiry_time/ts_am/365;
    float sigma = sqrt(del_t)*volatility; 
	
	int k = 0;
	int nt = blockDim.x;
	int start_index = bid*nt*ts_am;

	// 19 float/int register variables so far
	
	while ( k < ts_am ) 
	{
		// NOTE: (k < oeb) should be a good stopping criteria, stop computation as soon as
		// you exercise, but can lead to highly divergent code paths.  
		
		int index = start_index + k*nt + tid;
		spot_price = spot_price*exp(drift*del_t + sigma*norm_sample[index]);
		
		//if (index > width*height)
		//	cuPrintf ("index = %d for start_index = %d, k = %d, nt = %d, tid= %d\n", index, start_index, k, nt, tid );
		
		put_value = fmaxf( strike_price - spot_price, 0.0); //put
		
		//=======================Black-scholes continuation value========================//
		int kt = k+1;
		time = kt*del_t; 			// is the current time
		
		//if ( pathid == 256 )
		//	cuPrintf ("At t = %g, put value = %g; spot_price = %g, strike_price = %g, index = %d and normrand = %g\n",
		//	time, put_value, spot_price, strike_price, index, norm_sample[index]);
		
		ttm = (expiry_time - time)/365;
		d1 = log(spot_price/strike_price) + ( discount_rate + 0.5*volatility*volatility )*ttm;
		d2 = log(spot_price/strike_price) + ( discount_rate - 0.5*volatility*volatility )*ttm;
		den = volatility*sqrtf( ttm );
		d1 = d1/den;
		d2 = d2/den;

		h = strike_price*exp(-1*discount_rate*ttm)*phi(-1*d2) - spot_price*phi(-1*d1);
		//===============================================================================//
		if ( oeb > kt & put_value > h )
		{
		    oeb = kt;
		    cf_am = fmaxf( strike_price - spot_price, 0.0);
		}
		
		k++;
		//if ( pathid == 0 )
		//	cuPrintf ("----------------------\n");
	}
	
	
    option_value_eu[ pathid ] = put_value*exp(-1*discount_rate*expiry_time/365 );
	option_value_am[ pathid ] = cf_am*exp(-1*discount_rate*oeb*del_t ); 
	
}

// incomplete
static __global__ void generate_asset_price_paths_and_cash_flow_reg_mp(float *option_value_eu, float *option_value_am, int width, int height, InputData indata, float *norm_sample)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;

	int ts_am = height-1;
	
	// the following should be single read and broadcast for all threads I hope
	float spot_price = indata.S_0;
	float strike_price = indata.strike_price;
	float expiry_time = indata.expiry_time;
	float discount_rate = indata.discount_rate;
	float volatility = indata.volatility;
	
	//if ( pathid == 256  )
	//		cuPrintf ("\nAt t=0, spot_price = %g, strike_price = %g\n", spot_price, strike_price);
			
	// for decicing optimal exercise boundary in american options
	float put_value = 0;
	float h = 0;
	float time = 0;
	float d1, d2, den, ttm;
	
	// assuming uniformaly distributed option exercise times
	float drift = discount_rate - indata.dividend - 0.5*pow(volatility,2);
    float del_t = expiry_time/ts_am/365;
    float sigma = sqrt(del_t)*volatility; 
	
	int m = 1;
	int nt = blockDim.x;
	int start_index = 4*bid*nt*ts_am;
	int pathid = 4*blockDim.x * blockIdx.x + threadIdx.x;
	
	
	// 19 float/int register variables so far
	while( m <= 4)
	{
		int oeb = ts_am;		// optimal_exercise_boundary for this path
		float cf_am = 0;		// cash flow of american option for this path
		
		int k = 0;

		while ( k < ts_am ) 
		{
			// NOTE: (k < oeb) should be a good stopping criteria, stop computation as soon as
			// you exercise, but can lead to highly divergent code paths.  
		
			int index = start_index + k*nt + tid;
			spot_price = spot_price*exp(drift*del_t + sigma*norm_sample[index]);
		
			//if (pathid == 100000)
			//	cuPrintf ("pathid = %d for start_index = %d, k = %d, nt = %d, tid= %d, m = %d\n", pathid, start_index, k, nt, tid, m );
		
			put_value = fmaxf( strike_price - spot_price, 0.0); //put
		
			//=======================Black-scholes continuation value========================//
			int kt = k+1;
			time = kt*del_t; 			// is the current time
		
			//if ( tid == 0 || tid == 255)
			//	cuPrintf ("At t = %g, put value = %g; spot_price = %g, strike_price = %g, index = %d and normrand = %g\n",
			//	time, put_value, spot_price, strike_price, index, norm_sample[index]);
		
			ttm = (expiry_time - time)/365;
			d1 = log(spot_price/strike_price) + ( discount_rate + 0.5*volatility*volatility )*ttm;
			d2 = log(spot_price/strike_price) + ( discount_rate - 0.5*volatility*volatility )*ttm;
			den = volatility*sqrtf( ttm );
			d1 = d1/den;
			d2 = d2/den;

			h = strike_price*exp(-1*discount_rate*ttm)*phi(-1*d2) - spot_price*phi(-1*d1);
			//===============================================================================//
			if ( oeb > kt & put_value > h )
			{
				oeb = kt;
				cf_am = fmaxf( strike_price - spot_price, 0.0);
			}
		
			k++;
		}
		
		
		//if ( tid == 0 )
		//	cuPrintf ("----------------------\n");
		m++;
		
		option_value_eu[ pathid ] = put_value*exp(-1*discount_rate*expiry_time/365 );
		option_value_am[ pathid ] = cf_am*exp(-1*discount_rate*oeb*del_t ); 
		
		start_index = start_index + nt*ts_am;
		pathid = pathid + nt;
	}
	
    
	
}

static __global__ void generate_asset_price_paths_and_cash_flow(float *S, float *cash_flow, float *option_value, int width, int height, InputData indata, float *norm_sample) {

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    //InputData indata = inputdata;
    // shared memory to make sure accesses are fast (not sure if no use of tid affects things)

    //hiprandState state;

    //hiprand_init(indata.random_seed, tid, 0, &state);

    float drift = indata.discount_rate - indata.dividend - 0.5*pow(indata.volatility,2);
    float del_t = indata.expiry_time/(height-1)/365;
    float sigma = sqrt(del_t)*indata.volatility;
    S[tid*height] = indata.S_0;
    float temp = indata.S_0;
    for (int j = 1; j < height; j++ )
    {
	    S[tid*height+j] = temp = temp*exp(drift*del_t + sigma*norm_sample[tid*height+j]);
    }
    
    int expiry_index = height-1;
    // at the expiry time, the only choice is to exercise the option
    float discount_eu = exp(-1*indata.discount_rate*indata.expiry_time/365 );
	float cash_temp;
    cash_flow[tid] = cash_temp = fmaxf(indata.strike_price - S[tid*height+expiry_index], 0.0); //put
    option_value[tid] = cash_temp*discount_eu;

    //__syncthreads();

}

static __global__ void find_optimal_exercise_boundary_gpu(float *S, float *cash_flow, float *option_value, int width, int height,
														  InputData indata, float *x, float *h, int *optimal_exercise_boundary, float *cash_flow_am) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    int expiry_index = height-1;

    //InputData indata = inputdata;

    float del_t = indata.expiry_time/(height-1)/365;
    // discount for merican counterpart
    float discount = exp(-1*indata.discount_rate*del_t );

    float put_value = 0;
	
	optimal_exercise_boundary[tid] = expiry_index;
	float xtemp = 0;
	float htemp = 0;
    // for all other times when the option can be exercised, we comapre the
    // value of exercising and continuation value to find optimal exercise boundary  
    for ( int time = expiry_index-1; time >= 1; time-- ) // move back in time
    {
        put_value = fmaxf( indata.strike_price - S[tid*height+time], 0.0); //put

        xtemp = S[tid*height+time];
        cash_flow[tid] = put_value;

        htemp = get_black_scholes_continuation_value_gpu(xtemp, time, height, indata);

        if ( cash_flow[tid] > htemp )
        {
            optimal_exercise_boundary[tid] = time;
            cash_flow_am[tid] = fmaxf(indata.strike_price - S[tid*height+time], 0.0);
        }
    }
	
	
    //cash_flow_am[tid] = fmaxf(indata.strike_price - S[tid*height+optimal_exercise_boundary[tid]], 0.0); 
    //cash_flow_am[tid] = 0;
    discount = exp(-1*indata.discount_rate*optimal_exercise_boundary[tid]*del_t );
    option_value[tid] = cash_flow_am[tid]*discount;//*/ 

}

static __global__ void generate_asset_price_paths_and_cash_flow_multiple_paths(float *S, float *cash_flow, float *option_value, int width, int height, InputData indata, float *norm_sample) {

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    float drift = indata.discount_rate - indata.dividend - 0.5*pow(indata.volatility,2);
    float del_t = indata.expiry_time/(height-1)/365;
    float sigma = sqrt(del_t)*indata.volatility;
    S[tid*height] = indata.S_0;
    float temp  = indata.S_0;
    float temp1 = temp;
    float temp2 = temp;
    float temp3 = temp;
    for (int j = 1; j < height; j++ )
    {
	    S[(4*tid  )*height+j] = temp  = temp *exp(drift*del_t + sigma*norm_sample[(4*tid  )*height+j]);
	    S[(4*tid+1)*height+j] = temp1 = temp1*exp(drift*del_t + sigma*norm_sample[(4*tid+1)*height+j]);
	    S[(4*tid+2)*height+j] = temp2 = temp2*exp(drift*del_t + sigma*norm_sample[(4*tid+2)*height+j]);
	    S[(4*tid+3)*height+j] = temp3 = temp3*exp(drift*del_t + sigma*norm_sample[(4*tid+3)*height+j]);
    }
    
    int expiry_index = height-1;
  
    // at the expiry time, the only choice is to exercise the option
    float discount_eu = exp(-1*indata.discount_rate*indata.expiry_time/365 );
	float cash_temp;
    cash_flow[(4*tid)] = cash_temp = fmaxf(indata.strike_price - S[(4*tid  )*height+expiry_index], 0.0); //put
    option_value[(4*tid)] = cash_temp*discount_eu;
    
    cash_flow[(4*tid+1)] = cash_temp = fmaxf(indata.strike_price - S[(4*tid+1)*height+expiry_index], 0.0); //put
    option_value[(4*tid+1)] = cash_temp*discount_eu;
    
    cash_flow[(4*tid+2)] = cash_temp = fmaxf(indata.strike_price - S[(4*tid+2)*height+expiry_index], 0.0); //put
    option_value[(4*tid+2)] = cash_temp*discount_eu;
    
    cash_flow[(4*tid+3)] = cash_temp = fmaxf(indata.strike_price - S[(4*tid+3)*height+expiry_index], 0.0); //put
    option_value[(4*tid+3)] = cash_temp*discount_eu;

}

void checkError(hipError_t err) {

    if (err != hipSuccess) {
        fprintf(stderr, "cuda function failed (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}

//Main computations
extern "C" void generate_and_find_exercise_boundary()
{
	printf( "\nGPU COMPUTATION\n=============================\n");
	
    InputData h_indata;
    // read the input file for options relating to the number of paths, number
    // of discrete time-steps etc. 
    FileIO fileIO;
    fileIO.readInputFile((char*)"./input/options.txt", h_indata);

    float GPU_t = 0;
    // allocate memory to store all Monte Carlo paths, and intialize
    // the initial value of the asset at t=0.
    int num_paths = (h_indata.num_paths%2 == 0)?h_indata.num_paths:h_indata.num_paths+1;  

    float *d_S = NULL;
    float *d_x = NULL;
    float *d_h = NULL;
    float *d_cash_flow = NULL;
    float *d_option_value = NULL;
    float *d_option_value_am = NULL;
    float *d_cash_flow_am = NULL;
    
    int *d_optimal_exercise_boundary = NULL;
    
    int width = num_paths;
    int height = h_indata.num_time_steps+1;
    //size_t size = num_paths*(h_indata.num_time_steps+1)*sizeof(float);
    //printf("width=%d\n", width);

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
	
	hipEvent_t startt, stopt;
    hipEventCreate(&startt);
    hipEventCreate(&stopt);
    hipEventRecord(startt,0);
    
    checkError(hipMalloc((void**)&d_S, width*sizeof(float)*height));
    checkError(hipMalloc((void**)&d_x, width*sizeof(float)));
    checkError(hipMalloc((void**)&d_h, width*sizeof(float)));
    checkError(hipMalloc((void**)&d_cash_flow, width*sizeof(float)));
    checkError(hipMalloc((void**)&d_option_value, width*sizeof(float)));
    checkError(hipMalloc((void**)&d_option_value_am, width*sizeof(float)));
    checkError(hipMalloc((void**)&d_cash_flow_am, width*sizeof(float)));
    checkError(hipMalloc((void**)&d_optimal_exercise_boundary, width*sizeof(int)));
	
	hipEventRecord(stopt,0);
    hipEventSynchronize(stopt);
    hipEventElapsedTime(&GPU_t, startt, stopt);
	printf("\n### GPU: Time to cudamalloc: %fs\n", GPU_t/1000);
	
    int threadsPerBlock = 256;
    int blocksPerGrid = (int)ceil( 0.25*width/threadsPerBlock);

    printf("	- Blocks per Grid = %d\n", blocksPerGrid);
    printf("	- Threads per Block = %d\n", threadsPerBlock);
    
    random_normal normrnd;
    normrnd.zigset( 78542121 );

	size_t size_norm = width*height*sizeof(float);
    float *h_norm_sample = (float *) malloc(size_norm);

    /*for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {    
            h_norm_sample[i*height+j] = normrnd.RNOR();
            //printf("h = %f\n", h_norm_sample[i*height+j]);
        }
    }*/
    
    for (int j = 0; j < height; j++) {
    	for (int i = 0; i < width; i++) {    
            h_norm_sample[j*width + i] = normrnd.RNOR();
            //printf("h = %f\n", h_norm_sample[i*height+j]);
        }
    }
	printf("	- size of d_norm_sample: %d\n", size_norm/4);
	
    float *d_norm_sample = NULL;

    checkError(hipMalloc((void**)&d_norm_sample, size_norm));

    checkError(hipMemcpy(d_norm_sample, h_norm_sample, size_norm, hipMemcpyHostToDevice));
	
    cudaPrintfInit();
	
	hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2,0);
    //generate_asset_price_paths_and_cash_flow_reg<<<blocksPerGrid,threadsPerBlock>>>(d_option_value, d_option_value_am, width, height, h_indata, d_norm_sample);
    generate_asset_price_paths_and_cash_flow_reg_mp<<<blocksPerGrid,threadsPerBlock>>>(d_option_value, d_option_value_am, width, height, h_indata, d_norm_sample);
    //generate_asset_price_paths_and_cash_flow<<<blocksPerGrid,threadsPerBlock>>>(d_S, d_cash_flow, d_option_value, width, height, h_indata, d_norm_sample);
    //generate_asset_price_paths_and_cash_flow_multiple_paths<<<blocksPerGrid/4,threadsPerBlock>>>(d_S, d_cash_flow, d_option_value, width, height, h_indata, d_norm_sample);     
	
	hipEventRecord(stop2,0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&GPU_t, start2, stop2);
	printf("\n### GPU: Time to generate price paths: %fs\n", GPU_t/1000);
	
    thrust::device_ptr<float> dev_option_value_b(d_option_value);
    thrust::device_ptr<float> dev_option_value_e = dev_option_value_b + width;
    float sum = thrust::reduce(dev_option_value_b, dev_option_value_e, (float)0, thrust::plus<float>());
    float var_eu = thrust::transform_reduce(dev_option_value_b, dev_option_value_e, square<float>(), (float)0, thrust::plus<float>());


    float european_option_value  = sum/width;
    var_eu = (var_eu - pow(european_option_value, 2) )/width;

	hipEvent_t start3, stop3;
    hipEventCreate(&start3);
    hipEventCreate(&stop3);
    hipEventRecord(start3,0);
    
    //find_optimal_exercise_boundary_gpu<<<blocksPerGrid, threadsPerBlock>>>(d_S, d_cash_flow, d_option_value, width, height, h_indata, d_x, d_h, d_optimal_exercise_boundary, d_cash_flow_am);

	hipEventRecord(stop3,0);
    hipEventSynchronize(stop3);
    hipEventElapsedTime(&GPU_t, start3, stop3);
	printf("\n### Time to generate optimal exercise boundary: %fs\n", GPU_t/1000);
	
	thrust::device_ptr<float> dev_option_value_am_b(d_option_value_am);
    thrust::device_ptr<float> dev_option_value_am_e = dev_option_value_am_b + width;
    float sum_a = thrust::reduce(dev_option_value_am_b, dev_option_value_am_e, (float)0, thrust::plus<float>());
    float var_am = thrust::transform_reduce(dev_option_value_am_b, dev_option_value_am_e, square<float>(), (float)0, thrust::plus<float>());
    float american_option_value  = sum_a/width;
    var_am = (var_am - pow(american_option_value, 2) )/width;


    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&GPU_t, start, stop);

	// show memory usage of GPU
    size_t free_byte ;
    size_t total_byte ;
    
	hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte );

    if ( hipSuccess != cuda_status )
    {
	    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
     	//exit(1);
     }

     //double free_db = (double)free_byte ;
     //double total_db = (double)total_byte ;
     //double used_db = total_db - free_db ;

	printf("\n\nSUMMARY RESULTS FOR GPU\n------------------------\n");
	printf(" i) American Option:\n");
	printf("%40s:   %.6f \n", "Valuation at t=0", american_option_value);
	printf("%40s:   %.6f \n", "Std dev of the samples", sqrt(var_am) );
	float delta_am = 1.96*sqrt(var_am/width)/american_option_value;
	printf("%40s:   %.3g %% (w.r.t. true mean)\n", "Maximum rel error (95% confidence)", 100*delta_am/(1-delta_am) );
	printf("\nii) European Option:\n");
	printf("%40s:   %.6f \n", "Valuation at t=0", european_option_value);
	printf("%40s:   %.6f \n", "Std dev of the samples", sqrt(var_eu) );
	float delta_eu = 1.96*sqrt(var_eu/width)/european_option_value;
	printf("%40s:   %.3g %% (w.r.t. true mean)\n", "Maximum rel error (95% confidence)", 100*delta_eu/(1-delta_eu) );
	printf("\niii) Early Exercise Value: %g\n", american_option_value - european_option_value);
	
	printf("\n\nRESOURCE USAGE FOR GPU\n------------------------\n");
    printf("%40s: %.3fs\n", "Time in GPU",GPU_t/1000);
    printf("%40s: %.2f megabyte\n", "GPU memory estimate", (total_byte - free_byte)*9.53674e-7);

    cudaPrintfDisplay(stdout,true);
    cudaPrintfEnd();
    /*
       for (int i = 0; i < width; i++) {
    //for (int j = 0; j < height; j++) {    
    //printf("s[%d][%d] = %f\n", i,j,h_S[i*height+j]);
    printf("cashflow[%d] = %f\n", i,h_cash_flow[i]);
    printf("option_value[%d] = %f\n", i,h_option_value[i]);
    printf("var_eu[%d] = %f\n", i,h_var_eu[i]);
    //}
    }
     */

    checkError(hipFree(d_S));
    checkError(hipFree(d_x));
    checkError(hipFree(d_h));
    checkError(hipFree(d_cash_flow));
    checkError(hipFree(d_option_value));
    checkError(hipFree(d_cash_flow_am));
    checkError(hipFree(d_optimal_exercise_boundary));
}
